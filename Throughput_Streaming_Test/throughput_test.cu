/**
* High-Throughput Streaming Example 
* 
* This example shows a basic example of Peer-to-Peer DMA transfers between
* an NI FlexRIO module and a NVIDIA GPU that is capable of NVIDIA GPU Direct
* technology. This example will measure throughput between the two devices and
* report the measurement back to the user.
*
* For more information on NI FPGA functions, see the NI FPGA Interface C API 
* Help. For more information on NVIDIA CUDA functions and operation, see the
* help files included with the NVIDIA CUDA Driver.
*
* Date:         12/14/2016
* Author:       John Gentile
*/

#include <stdio.h>
#include <sys/time.h>
#include <sys/times.h>
#include "NiFpga_FPGA_main.h"

#define NX              (128*8192) //4096 (64k pages)
#define NFRAMES         3
#define BATCH_NFRAMES   1000
#define CUDA_NTHREADS   1024
#define CUDA_NBLOCKS    (NX/CUDA_NTHREADS)

// use inline definition for error checking to allow easy app exit
#define CHECKSTAT(stat) if (stat != 0) { printf("%d: Error: %d\n", __LINE__, stat); return 1; }

// keep datatypes uniform between FIFOs and operations
typedef uint64_t fifotype;


int main(int argc, char **argv)
{
  // initialize NI FPGA interfaces; use status variable for error handling
  printf("Initializing NI FPGA...\n");
  CHECKSTAT(NiFpga_Initialize());
  NiFpga_Session session;
  
  // Download bitfile to target; get path to bitfile
  // TODO: change to full path to bitfile as necessary
  CHECKSTAT(NiFpga_Open("/home/nitest/FlexRIO-P2P-GPU/Throughput_Streaming_Test/NiFpga_FPGA_main.lvbitx", NiFpga_FPGA_main_Signature, "RIO0", 0, &session));
  
  // Allocate CUDA memory; the CUDA device will operate on frames of samples so
  // we willl allocate two frames worth of space 
  printf("Allocating CUDA Memory: ");
  fifotype *gpu_mem;
  hipError_t cuerr = hipMalloc(&gpu_mem, sizeof(fifotype)*NX*NFRAMES);
  printf("%p\n", gpu_mem);

  // Configure P2P FIFO between FlexRIO and GPU using NVIDIA GPU Direct
  CHECKSTAT(NiFpga_ConfigureFifoBuffer(session, NiFpga_FPGA_main_TargetToHostFifoU64_FlexRIO_FIFO, (uint64_t)gpu_mem, NX*NFRAMES, NULL, NiFpga_DmaBufferType_NvidiaGpuDirectRdma)); 
  
  CHECKSTAT(NiFpga_StartFifo(session, NiFpga_FPGA_main_TargetToHostFifoU64_FlexRIO_FIFO));
  
  // Setup batch size to be very large
  NiFpga_WriteU64(session, NiFpga_FPGA_main_ControlU64_BatchSize, (long long)NX*(long long)BATCH_NFRAMES+1000);

  // Start transferring data
  printf("Transferring Data");
  NiFpga_WriteBool(session, NiFpga_FPGA_main_ControlBool_StartTransfer, NiFpga_True);

  size_t tot_bytes = 0; // total bytes transferred
  struct timeval start_time;
  gettimeofday(&start_time, NULL);

  uint64_t running_count = 0;
  fifotype *datap;
  size_t elems_acquired, elems_remaining = NX;

  // Start recording the user/system times
  struct tms start_tms, end_tms;
  times(&start_tms);

  int i;
  for (i=0; i<BATCH_NFRAMES; i++)
  {
    // Acquire data from FlexRIO -> GPU
    tot_bytes += sizeof(fifotype)*(NX);
    CHECKSTAT(NiFpga_AcquireFifoReadElementsU64(session, NiFpga_FPGA_main_TargetToHostFifoU64_FlexRIO_FIFO, &datap, NX, 3000, &elems_acquired, &elems_remaining));
    

  
  // Close NI FPGA References; must be last NiFpga calls
  printf("Stopping NI FPGA...\n");
  CHECKSTAT(NiFpga_StopFifo(session, NiFpga_HighThroughputStreamingFPGAPXIe797xR_TargetToHostFifoI16_InputFIFO));
  CHECKSTAT(NiFpga_Close(session, 0));
  CHECKSTAT(NiFpga_Finalize());

  return 0;

}
